
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 32
#define BLOCK_WIDTH 3
__global__ void hello()
{
printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
//printf("Hello world! I'm thread %d\n", threadIdx.x);
}int main(int argc,char **argv)
{
// launch the kernel
hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
// force the printf()s to flush
hipDeviceSynchronize();
printf("That's all!\n");
return 0;
}